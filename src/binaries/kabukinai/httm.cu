#include "hip/hip_runtime.h"
#include "kabukinai.h"
#include "hiprand/hiprand_kernel.h"
#include "math.h"

// A kernel to extract slices in reverse, 'cause there's not a hipMemcpy variant for that
// Launch with a grid dimension of the image height, block dimension of the slice image width

__global__ void extract_reverse_floating_slice(
	float *raster, 
	float *slice, 
	int slice_width, 
	int image_width 
) 
{
	slice[slice_width * blockIdx.x + blockDim.x - 1 - threadIdx.x] = 
		raster[image_width * blockIdx.x + threadIdx.x];
}

// Replace a floating point simulated image with a sliced version
__host__ void
to_slices( simulation_data *d ) {
	float * slices;
	int height = d->dimensions[0];
	int width = d->dimensions[1];
	int slice_image_width = width/d->number_of_slices;
	int slice_width = slice_image_width + d->early_dark_pixels + d->late_dark_pixels;
	int slice_height = height + d->smear_rows + d->final_dark_rows;
	int slice_size = slice_width*slice_height;
	
	PANIC_ON_BAD_CUDA_STATUS(hipMalloc((void **) &slices, slice_size*d->number_of_slices*sizeof(float)));
		
	for( int s = 0; s < d->number_of_slices; s += 1) {
		float *this_slice = slices + s * slice_size;
		PANIC_ON_BAD_CUDA_STATUS(
			hipMemset2DAsync( // zero early darks
				this_slice, 
				slice_width * sizeof(float), 
				0, 
				d->early_dark_pixels * sizeof(float), 
				slice_height ));
		PANIC_ON_BAD_CUDA_STATUS(
			hipMemset2DAsync( // zero late darks
				this_slice + d->early_dark_pixels + slice_image_width, 
				slice_width * sizeof(float), 
				0, 
				d->late_dark_pixels * sizeof(float), 
				slice_height ));
		PANIC_ON_BAD_CUDA_STATUS(
			hipMemset2DAsync( // sero smear and final darks
				this_slice + height * slice_width, 
				slice_width * sizeof(float), 
				0, 
				slice_width * sizeof(float), 
				d->smear_rows + d->final_dark_rows ));
		
		if( s & 1 ) {	// flip odd slices
			extract_reverse_floating_slice<<<height, slice_image_width>>>( 
				d->image_pixels + s * slice_image_width, 
				this_slice + d->early_dark_pixels,
				slice_width, width );
		} else {
			PANIC_ON_BAD_CUDA_STATUS(		
				hipMemcpy2DAsync( 
					this_slice + d->early_dark_pixels,
					slice_width * sizeof(float),
					d->image_pixels + s * slice_image_width,
					width * sizeof(float),
					slice_image_width * sizeof(float),
					height,
					hipMemcpyDeviceToDevice
			));			
		}
	}
	
	PANIC_ON_BAD_CUDA_STATUS(hipDeviceSynchronize());
	PANIC_ON_BAD_CUDA_STATUS(hipFree(d->image_pixels));
	d->image_pixels = slices;
}



__global__ void cu_smear( 
	float * slices, 
	int slice_size, 
	int early_darks, 
	int smear_rows, 
	int image_height, 
	int slice_width,
	float smear_ratio
)
{
	double smear = 0.0;
	float *image_pixel = slices + early_darks + blockIdx.x * slice_size + threadIdx.x;
	for( int i = 0; i < image_height; i += 1) {
		smear += *image_pixel;
		image_pixel += slice_width;
	}
	// Note that image_pixel automagically winds up pointing to the first smear row
	
	smear *= smear_ratio;
	
	for( int i = 0; i < smear_rows; i += 1 ) {
		*image_pixel = smear;
		image_pixel += slice_width;
	}
}


__host__ void add_smear( simulation_data *d ) {
	int height = d->dimensions[0];
	int width = d->dimensions[1];
	int slice_image_width = width/d->number_of_slices;
	int slice_width = slice_image_width + d->early_dark_pixels + d->late_dark_pixels;
	int slice_height = height + d->smear_rows + d->final_dark_rows;
	int slice_size = slice_width*slice_height;
	
	cu_smear<<<d->number_of_slices, slice_image_width>>>( 
		d->image_pixels, 
		slice_size, 
		d->early_dark_pixels,
		d->smear_rows,
		height,
		slice_width,
		d->smear_ratio );
		
	PANIC_ON_BAD_CUDA_STATUS(hipDeviceSynchronize());
}


__global__ void cu_noise( 
	float * slice, 
	int slice_size, 
	int height, 
	int slice_width,
	unsigned long long random_seed,
	unsigned long long random_offset,
	float readout_noise_variance
)
{
	hiprandState_t random_state;
	hiprand_init ( 
		random_seed, 
		(unsigned long long) threadIdx.x, 
		random_offset, 
		&random_state );

	float *image_pixel = slice + threadIdx.x;
	for( int i = 0; i < height; i += 1) {
		
		*image_pixel += 
			sqrtf(readout_noise_variance + *image_pixel) * 
			hiprand_normal (&random_state);
		image_pixel += slice_width;
	}
}


__host__ void add_noise( simulation_data *d ) {
	int height = d->dimensions[0];
	int width = d->dimensions[1];
	int slice_image_width = width/d->number_of_slices;
	int slice_width = slice_image_width + d->early_dark_pixels + d->late_dark_pixels;
	int slice_height = height + d->smear_rows + d->final_dark_rows;
	int slice_size = slice_width*slice_height;
	
	for( int s = 0; s < d->number_of_slices; s += 1 ) {
	
		cu_noise<<<1, slice_image_width>>>( 
			d->image_pixels + s * slice_size, 
			slice_size, 
			height,
			slice_width,
			d->random_seed,
			d->random_offset + s * 1000000,
			d->read_noise_variance[s]
		);
	}
		
	PANIC_ON_BAD_CUDA_STATUS(hipDeviceSynchronize());
	
	d->random_offset += 10000000;	// be sure we get new numbers next call
}

