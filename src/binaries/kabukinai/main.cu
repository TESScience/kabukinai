#include <stdio.h>
#include "psf.h"
#include "sum_intensities.cu"
#include "kabukinai.h"
#include "star_data.h"
#include <stdlib.h>
#include <stddef.h>

#define PANEL_SIDE_DIMENSION 32
#define TESS_IMAGE_HEIGHT 2058
#define TESS_IMAGE_WIDTH 2048

int main(const int argc, char * argv[])
{
	psf_data point_spread_function_data;
        read_psf_fits(&point_spread_function_data, argv[1]);

        setup_psf_texture(point_spread_function_data.dimensions[0], 
			  point_spread_function_data.dimensions[1], 
			  point_spread_function_data.image_pixels);

        star_data star_data_from_file;
        dimensions image_dimensions;
        image_dimensions.x_dimension = TESS_IMAGE_WIDTH;
        image_dimensions.y_dimension = TESS_IMAGE_HEIGHT;

	dimensions single_panel_pixel_dimensions;
        single_panel_pixel_dimensions.x_dimension = PANEL_SIDE_DIMENSION;
        single_panel_pixel_dimensions.y_dimension = PANEL_SIDE_DIMENSION;

	parse_star_data_from_tsv(&star_data_from_file, argv[2], image_dimensions, single_panel_pixel_dimensions);

	const size_t image_size = sizeof(float) * image_dimensions.x_dimension * image_dimensions.y_dimension;

	float * pixels;
	PANIC_ON_BAD_CUDA_STATUS(hipMalloc((void**)&pixels, image_size));
        PANIC_ON_BAD_CUDA_STATUS(hipMemset((void**)&pixels, 0, image_size));
        
	star * stars;
	const size_t stars_size = sizeof(star) * number_of_stars(star_data_from_file);
	PANIC_ON_BAD_CUDA_STATUS(hipMalloc((void**)&stars, stars_size));
	PANIC_ON_BAD_CUDA_STATUS(hipMemcpy(stars, star_data_from_file.stars, stars_size, hipMemcpyHostToDevice));

	int * panel_indices;
	const size_t panel_indices_size = sizeof(int) * number_of_panel_indices(star_data_from_file);
	PANIC_ON_BAD_CUDA_STATUS(hipMalloc((void**)&panel_indices, panel_indices_size));
	PANIC_ON_BAD_CUDA_STATUS(hipMemcpy(
				panel_indices, 
				star_data_from_file.panel_indices, 
				panel_indices_size, 
				hipMemcpyHostToDevice));

	dim3 dgrid(image_dimensions.y_dimension/single_panel_pixel_dimensions.y_dimension,
			image_dimensions.x_dimension/single_panel_pixel_dimensions.x_dimension);
	dim3 dblock(single_panel_pixel_dimensions.y_dimension, single_panel_pixel_dimensions.x_dimension);
	sum_intensities_for_pixel<<<dgrid, dblock>>>(pixels, stars, panel_indices, star_data_from_file.meta_data);


 	PANIC_ON_BAD_CUDA_STATUS(hipDeviceSynchronize());
 
        /*
         * Copy the result back to the host.
         */

	simulation_data result;
	result.image_pixels = (float *) malloc(image_size);
        result.dimensions[0] = image_dimensions.x_dimension;
	result.dimensions[1] = image_dimensions.y_dimension;
	PANIC_ON_BAD_CUDA_STATUS(hipMemcpy(result.image_pixels, pixels, image_size, hipMemcpyDeviceToHost));

        
        write_simulation_fits(result, argv[3], "TODO"); 
 	
 	PANIC_ON_BAD_CUDA_STATUS(hipDeviceReset());
	psf_data_release(point_spread_function_data);
	simulation_data_release(result);
	exit(EXIT_SUCCESS);
}
