#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * TODO make comment regarding matrix orientation versus image coordinates.
 */

/*
 * Texture reference. File scope, mutable, shared between device
 * and host. Holds details of the texture. The stuff in <> is immutable.
 * Other details are mutable, and are set in host code, including
 * its binding to a data array.
 */

texture<float, hipTextureType2D, hipReadModeElementType> myTexture;

__global__ void try_interpolation( float *output ) {

/*
 * Distort our image to test interpolation and clipping.
 */
	
	float y = threadIdx.y * 0.75 - 0.5;
	float x = threadIdx.x *0.5;
	
	output[ threadIdx.y * blockDim.x + threadIdx.x ] =
		tex2D( myTexture, y, x );
}

int main()
{
	hipError_t code;

/*
 * Make a 2d test pattern as an ordinary C array.
 */
 
	const int width = 4 , height = 4 ;

	float data[height][width] ;
	for ( int y =0 ; y<height ; y++ ) {
			for ( int x = 0 ; x<width; x++ ){
			data[y][x] = x*x+y*y;
		}
	}
	const int size = width*height*sizeof(float) ;

/*
 * Print the contents of the input array.
 */	
	
	for ( int y =0 ; y<height ; y++ ) {
			for ( int x = 0 ; x<width; x++ ){
			printf( "%10g ", data[y][x]);
		}
		printf( "\n");
	}
	printf( "\n" );

/*
 * A hipChannelFormatDesc is a structure that defines the
 * contents of an element of a hipArray. Such an element can contain
 * up to four numbers. hipCreateChannelDesc() allows you to specify the 
 * length of each of the four in bits (!) along with a type category
 * common to all. So, this complicated call defines an ordinaly scalar
 * float as an array element.
 */

/*
 * A hipChannelFormatDesc is a structure that defines the
 * contents of an element of a hipArray.
 * 
 * An element of a hipArray has up to four numbers of variable length.
 * 
 * struct hipChannelFormatDesc {
 *    int x, y, z, w;
 *    enum hipChannelFormatKind  f;
 * }

It is constructed with a function `hipCreateChannelDesc`
hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind  f) ;

Here x,y,z, and w are the number of bits for each dimension
*/
 
	const hipChannelFormatDesc floatDesc = 
		hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat );

/*
 * A hipArray is an array of objects defined by a hipChannelFormatDesc.
 * Allocate one of these on the device.
 */
 
	hipArray* floatArray;
	code = hipMallocArray(&floatArray, &floatDesc, height, width);
	if( code ) {
		printf( "hipMallocArray: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}

/*
 * Now copy our ordinary C array to the hipArray on the device.
 */

	code = hipMemcpyToArray(floatArray, 0, 0, data, size,
		hipMemcpyHostToDevice);
	if( code ) {
		printf( "hipMemcpyToArray: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}


/*
 * Now set up the mutable fields of the texture reference.
 *
 * First, set it up to yield zero for requests outside the array.
 */
 
	myTexture.addressMode[0] = hipAddressModeBorder;
	myTexture.addressMode[1] = hipAddressModeBorder;

/*
 * Interpolate between samples.
 */
	
	myTexture.filterMode = hipFilterModeLinear;

/*
 * Use [0,height-1][0, width-1] as the range for the floating 
 * coordinates.
 */
	
	myTexture.normalized = false;

/*
 * Finally, bind the texture to its data.
 */
	
	code = hipBindTextureToArray(myTexture, floatArray, floatDesc);
	if( code ) {
		printf( "hipBindTextureToArray: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}

/*
 * For no good reason, make the output array
 * the same size as the texture.
 */
	
	float* output;
	code =hipMalloc(&output, size );
	if( code ) {
		printf( "hipMalloc: %s\n", hipGetErrorString(code));
		exit( 1 );
	}
/*
 * Run the kernel. Note that we don't have to tell it sizes
 * of things in the args, as those are implied by the block dimensions.
 */
	dim3 blocks_dimension( height, width );
	try_interpolation<<< 1, blocks_dimension  >>>
		( output );
	code = hipDeviceSynchronize();
	if( code ) {
		printf( "hipDeviceSynchronize: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}

/*
 * Copy the result back to the host.
 */

	float result[height][width];
	code = hipMemcpy(result, output, size, hipMemcpyDeviceToHost);
	if( code ){
		printf( "hipMemcpyDeviceToHost: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}
/*
 * Print the result.
 */
	
	for ( int y =0 ; y<height ; y++ ) {
			for ( int x = 0 ; x<width; x++ ){
			printf( "%10g ", result[y][x]);
		}
		printf( "\n");
	}

/*
 * If you're really done, you can tidy up with a bulldozer ;-)
 */
	
	code = hipDeviceReset();
	if( code ) {
		printf( "hipMemcpyDeviceToHost: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}
}
