#include "hip/hip_runtime.h"
#include "../../libraries/star_data/star_data.h"
#include "stdbool.h"

/*
 * A block is associated with the panels that contain stars that
 * can influence it. If there are, e.g., 9 such panels, threads in
 * the block can access them by numbers 0-8. The functions below figure
 * this out using blockIdx.x and blockIdx.y to identify which block they
 * are serving.
 */

/*
 * Indicate whether the block is has an associated panel numbered i.
 */

__device__ inline bool block_has_panel( int i ){
	return false;	/* STUB */
}

/*
 * Return the number of stars in panel i.
 */

__device__ inline int number_of_stars_in_panel( int i ){
	return 0;	/* STUB */
}

/*
 * Return a pointer to the stars in panel i.
 */

__device__ inline star *array_of_stars_in_panel( int i ){
	return NULL;	/* STUB */
}

/*
 * The following use blockIdx and threadIdx to figure out which
 * pixel the thread represents.
 */
 
/* 
 * Get the PSF amplitude at the thread's pixel for the star at x, y.
 */

__device__ inline float cu_psf( float x, float y, int color ) {
	return 0.0;	/* STUB */
}

/*
 * Set the thread's pixel value in the output image.
 */

__device__ inline void set_pixel_in_raster( float my_pixel ){
	return;		/* STUB */
}


__global__ void sum_intensities_for_pixel( ) {
	
	float my_pixel = 0.0;	/* This thread's pixel value */
	int i, j, color;

	for( i = 0; block_has_panel( i ); i += 1){
		int number_of_stars = number_of_stars_in_panel( i );
		star * starp = array_of_stars_in_panel( i );
		for( j = 0; j < number_of_stars; j += 1 ){
			for( color = 0; color < STAR_COLORS; color += 1){
				my_pixel += starp->intensities[color] *
					cu_psf( starp->x, starp->y, color );
			}
		}
	}
	set_pixel_in_raster( my_pixel );
}
