#include "hip/hip_runtime.h"
#ifndef SUM_INTENSITIES_CU
#define SUM_INTENSITIES_CU
#include "star_data.h"
#include "kabukinai.h"
#include <stdio.h>
#include <stdlib.h>

// TODO: This isn't sanitary, we must use objects!
// Texture reference, local to this file
texture<float, hipTextureType2DLayered, hipReadModeElementType> psf_texture;

// Call this function to make the texture from a C data array

__host__ void setup_psf_texture(const int height, const int width, const float *data) {

	// Make the type definition for a single float element
	
	const hipChannelFormatDesc floatDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat );
	
	// Make an array to hold the texture on the device

	hipArray* floatArray;
	PANIC_ON_BAD_CUDA_STATUS(hipMalloc3DArray(&floatArray, &floatDesc, make_hipExtent( height, width, STAR_COLORS ), 0));

	// Copy the data to the array
	const size_t size = height * width * STAR_COLORS * sizeof( float );
	PANIC_ON_BAD_CUDA_STATUS(hipMemcpyToArray(floatArray, 0, 0, data, size, hipMemcpyHostToDevice));

	// Return zero for accesses outside the PSF texture
	
	psf_texture.addressMode[0] = hipAddressModeBorder;
	psf_texture.addressMode[1] = hipAddressModeBorder;
	
	// Interpolate between samples
	
	psf_texture.filterMode = hipFilterModeLinear;

	// Use [0,1] as the coordinate limits of the texture.
	// This means cu_psf need not pay attention to oversampling.
	
	psf_texture.normalized = true;

	// Bind the texture to its data
		
	PANIC_ON_BAD_CUDA_STATUS(hipBindTextureToArray(psf_texture, floatArray, floatDesc));
}

// Get the PSF amplitude at pixel coordinates x, y relative to the 
// center of the PSF

__device__ inline float cu_psf(const float x, const float y, const int color, const star_meta_data meta_data) {
    const float norm_x = x / meta_data.single_panel_pixel_dimensions.x_dimension + 0.5;
    const float norm_y = y / meta_data.single_panel_pixel_dimensions.y_dimension + 0.5;
    return tex2DLayered(psf_texture, norm_x, norm_y, color);
}


__global__ void
sum_intensities_for_pixel(float *pixel, const star *stars, int *panel_indices, const star_meta_data meta_data) {

    float my_pixel = 0.0;   // This thread's pixel value
    const int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int pixel_y = blockIdx.y * blockDim.y + threadIdx.x;
    const int pixel_index = pixel_y * meta_data.image_dimensions.x_dimension + pixel_x;

    for (int panel_indexX = blockIdx.x - 1; panel_indexX <= blockIdx.x + 1; ++panel_indexX) {
        for (int panel_indexY = blockIdx.y - 1; panel_indexY <= blockIdx.y + 1; ++panel_indexY) {
            const int neighborhood_index = panel_index_lookup(panel_indexX, panel_indexY, meta_data);
            const int panel_start = panel_indices[neighborhood_index];
            const int panel_end = panel_indices[neighborhood_index + 1];
            for (int star_index = panel_start; star_index < panel_end; ++star_index) {
                const star star_data = stars[star_index];
                for (int color = 0; color < STAR_COLORS; ++color)
                    my_pixel +=
                            star_data.intensities[color] *
                            cu_psf(star_data.point.x - pixel_x, star_data.point.y - pixel_y, color, meta_data);
            }
        }
    }
    pixel[pixel_index] = my_pixel;
}

#endif // SUM_INTENSITIES_CU
