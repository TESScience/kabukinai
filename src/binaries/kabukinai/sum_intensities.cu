#include "hip/hip_runtime.h"
#include "../../libraries/star_data/star_data.h"
#include <stdio.h>
#include <stdlib.h>

// Texture reference, local to this file

texture<float, hipTextureType2DLayered, hipReadModeElementType> psf_texture;

// Call this function to make the texture from a C data array

__host__ void setup_psf_texture( int height, int width, float *data) {
	
	hipError_t code;

	// Make the type definition for a single float element
	
	const hipChannelFormatDesc floatDesc = 
		hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat );
	
	// Make an array to hold the texture on the device

	hipArray* floatArray;
	code = hipMalloc3DArray(&floatArray, &floatDesc,
		 make_hipExtent( height, width, STAR_COLORS ), 0);
	if( code ) {
		fprintf( stderr, "hipMalloc3DArray: %s\n",
			hipGetErrorString(code));
			exit( 1 );
	}
	
	// Copy the data to the array
	
	size_t size = height * width * STAR_COLORS * sizeof( float );
	code = hipMemcpyToArray(floatArray, 0, 0, data, size,
		hipMemcpyHostToDevice);
	if( code ) {
		fprintf( stderr, "hipMemcpyToArray: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}
	
	// Return zero for accesses outside the PSF texture
	
	psf_texture.addressMode[0] = hipAddressModeBorder;
	psf_texture.addressMode[1] = hipAddressModeBorder;
	
	// Interpolate between samples
	
	psf_texture.filterMode = hipFilterModeLinear;

	// Use [0,1] as the coordinate limits of the texture.
	// This means cu_psf need not pay attention to oversampling.
	
	psf_texture.normalized = true;

	// Bind the texture to its data
		
	code = hipBindTextureToArray(psf_texture, floatArray, floatDesc);
	if( code ) {
		fprintf( stderr, "hipBindTextureToArray: %s\n",
			hipGetErrorString(code));
		exit( 1 );
	}
}

// Get the PSF amplitude at pixel coordinates x, y relative to the 
// center of the PSF

__device__ inline float cu_psf(float x, float y, int color) {
    float norm_x = x / blockDim.x + 0.5;
    float norm_y = y / blockDim.y + 0.5;
    return tex2DLayered(psf_texture, norm_x, norm_y, color);
}


__global__ void
sum_intensities_for_pixel(float *pixel, const star *stars, int *panel_indices, const star_meta_data meta_data) {

    float my_pixel = 0.0;   /* This thread's pixel value */
    const int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int pixel_y = blockIdx.y * blockDim.y + threadIdx.x;
    const int pixel_index = pixel_y * meta_data.image_dimensions.x_dimension + pixel_x;

    for (int panel_indexX = blockIdx.x - 1; panel_indexX <= blockIdx.x + 1; ++panel_indexX) {
        for (int panel_indexY = blockIdx.y - 1; panel_indexY <= blockIdx.y + 1; ++panel_indexY) {
            const int neighborhood_index = panel_index_lookup(panel_indexX, panel_indexY, meta_data);
            const int panel_start = panel_indices[neighborhood_index];
            const int panel_end = panel_indices[neighborhood_index + 1];
            for (int star_index = panel_start; star_index < panel_end; ++star_index) {
                const star star_data = stars[star_index];
                for (int color = 0; color < STAR_COLORS; ++color)
                    my_pixel +=
                            star_data.intensities[color] *
                            cu_psf(star_data.point.x - pixel_x, star_data.point.y - pixel_y, color);
            }
        }
    }
    pixel[pixel_index] = my_pixel;
}
